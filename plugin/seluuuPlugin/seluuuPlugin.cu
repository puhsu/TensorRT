#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


 #include "seluuuPlugin.h"
 #include <math.h>
 #include <hip/hip_fp16.h>


 template <typename T_DATA>
     __global__ void kernelCopy(
         int N,
         T_DATA* inputs,
         T_DATA* outputs
         )
 {
     int index = blockIdx.x * blockDim.x + threadIdx.x;
     if (index < N){
         outputs[index] = inputs[index];
     }
     __syncthreads();
 }

constexpr float alpha = 1.6732632423543772848170429916717f;
constexpr float beta = 1.0507009873554804934193349852946f;

__global__ void kernel_selu(
    int N,
    const float *inputs,
    float* outputs
)
 {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N){
         if (inputs[index] > 0) {
            outputs[index] = beta * inputs[index];
         } else {
            outputs[index] = beta * alpha * (exp(inputs[index]) - 1);
         }
    }
 }

 int inference(
     int batchSize,
     int dataDim,
     float* inputs,
     float* outputs,
     hipStream_t stream
) {
    int N = batchSize * dataDim;
    int N_blocks = N % 512 == 0 ? N / 512 : N / 512 + 1;

    kernel_selu<<<N_blocks, 512, 0, stream>>>(N, inputs, outputs);

     hipError_t err = hipGetLastError();
     if ( hipSuccess != err )
     {
         fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 __FILE__, __LINE__, hipGetErrorString( err ) );
         return 1;
     }
     return 0;
}

 int SeluuuPlugin::enqueue(
     int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream)
{
    return inference(batchSize, dataDim, (float*)inputs[0], (float*)outputs[0], stream);
}
